#include <vector>

#include "caffe/layers/base_data_layer.hpp"

namespace caffe {

template <typename Dtype>
void BasePrefetchingDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  //LOG(INFO)<<iter_idx_;
  if (iter_idx_==1){
	  Batch<Dtype>* batch = prefetch_full_.pop("Data layer prefetch queue empty");
	  // Reshape to loaded data.
	  top[0]->ReshapeLike(batch->data_);
	  // Copy the data
	  caffe_copy(batch->data_.count(), batch->data_.gpu_data(),
		  top[0]->mutable_gpu_data());
	  if (this->output_labels_) {
		// Reshape to loaded labels.
		top[1]->ReshapeLike(batch->label_);
		// Copy the labels.
		caffe_copy(batch->label_.count(), batch->label_.gpu_data(),
			top[1]->mutable_gpu_data());
	  }
	  // Ensure the copy is synchronous wrt the host, so that the next batch isn't
	  // copied in meanwhile.
	  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
	  prefetch_free_.push(batch);
  }
  iter_idx_ = iter_idx_ == 2 ? 1 : iter_idx_ + 1;
}

INSTANTIATE_LAYER_GPU_FORWARD(BasePrefetchingDataLayer);

}  // namespace caffe
